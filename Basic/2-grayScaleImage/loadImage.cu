
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <stdio.h>

using namespace cv;
int main() {
	Mat image = imread("lena512color.tiff", CV_LOAD_IMAGE_COLOR);
	if (image.empty())
	{
		printf("!!! Failed imread(): image not found\n");
		exit(-1);
	}
	printf("channels: %d\n", image.channels());
	printf("dims: %d\n", image.dims);
	printf("rows: %d\n", image.rows);
	printf("cols: %d\n", image.cols);
	namedWindow("Display window", CV_WINDOW_AUTOSIZE);// Create a window for display
	imshow("Display window", image);
	waitKey(0);
	return 0;
}