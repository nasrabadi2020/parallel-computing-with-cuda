#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

int main() {
    int numberOfDevices;
    hipGetDeviceCount(&numberOfDevices);
    printf("Number of CUDA-capable GPUs: %d\n", numberOfDevices);

    for (int i = 0; i < numberOfDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);

        printf("\n=================== Device %d ===================\n", i);
        printf("Name: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("Clock rate: %.0f MHz (%.2f GHz)\n", prop.clockRate * 1e-3f, prop.clockRate * 1e-6f);
        printf("Global memory: %.0f MB (%llu bytes)\n", prop.totalGlobalMem / 1048576.0f, (unsigned long long)prop.totalGlobalMem);
        printf("Shared memory per block: %zu bytes\n", (size_t)prop.sharedMemPerBlock);
        printf("Shared memory per multiprocessor: %zu bytes\n", (size_t)prop.sharedMemPerMultiprocessor);
        printf("Constant memory: %zu bytes\n", (size_t)prop.totalConstMem);
        printf("Registers per block: %d\n", prop.regsPerBlock);
        printf("Registers per multiprocessor: %d\n", prop.regsPerMultiprocessor);
        printf("Warp size: %d\n", prop.warpSize);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max threads per multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
        printf("Number of multiprocessors (SMs): %d\n", prop.multiProcessorCount);
        printf("Max block dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("Memory Clock Rate: %.0f MHz\n", prop.memoryClockRate * 1e-3f);
        printf("Memory Bus Width: %d-bit\n", prop.memoryBusWidth);
        printf("Texture Alignment: %zu bytes\n", (size_t)prop.textureAlignment);
        printf("Device Overlap (Async copy + kernel): %s\n", prop.deviceOverlap ? "Supported" : "Not supported");

        // ویژگی‌های پیشرفته‌تر:
        printf("Can map host memory: %s\n", prop.canMapHostMemory ? "Yes" : "No");
        printf("Concurrent kernel execution: %s\n", prop.concurrentKernels ? "Yes" : "No");
        printf("ECC enabled: %s\n", prop.ECCEnabled ? "Yes" : "No");
        printf("Integrated GPU (shared with CPU): %s\n", prop.integrated ? "Yes" : "No");
        printf("Multi-GPU board: %s\n", prop.isMultiGpuBoard ? "Yes" : "No");

        printf("PCI Bus ID: %d\n", prop.pciBusID);
        printf("PCI Device ID: %d\n", prop.pciDeviceID);
        printf("Async Engine Count: %d\n", prop.asyncEngineCount);
        printf("L2 Cache Size: %d bytes\n", prop.l2CacheSize);

        printf("Managed memory supported: %s\n", prop.managedMemory ? "Yes" : "No");
        printf("Pageable memory access: %s\n", prop.pageableMemoryAccess ? "Yes" : "No");
        printf("Concurrent managed access: %s\n", prop.concurrentManagedAccess ? "Yes" : "No");
    }

    return 0;
}
